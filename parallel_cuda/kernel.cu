#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "kernel.h"
#include <stdio.h>
#include <stdlib.h>

__global__ void classifyPointsKernel(Color4 *centroid, int *label, Color4 *pixels, int *migration_count, int cluster_count, int total_pixel){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < total_pixel){
        int index = -1;
        int min_dist = 10000000;
        for (int j = 0; j < cluster_count; j++)
        {
            int dist = (pixels[i].r - centroid[j].r) * (pixels[i].r - centroid[j].r)\
             + (pixels[i].g - centroid[j].g) * (pixels[i].g - centroid[j].g) + \
             (pixels[i].b - centroid[j].b) * (pixels[i].b - centroid[j].b) +\
              (pixels[i].a - centroid[j].a) * (pixels[i].a - centroid[j].a);
            //printf("%d ",pixels[i].r);
            if (dist < min_dist)
            {   
                index = j;
                min_dist = dist;
            }
        }
        // printf("%d\n",index);
        if (index != label[i])
            atomicAdd(migration_count, 1);
        label[i] = index;
    }
}

void host_classify_points(Color4 *centroid, int *label, Color4 *pixels, int *migration_count, int cluster_count, int total_pixel)
{
    Color4 *centroid_d;
    hipMalloc(&centroid_d, cluster_count*sizeof(Color4));
    hipMemcpy(centroid_d, centroid, cluster_count*sizeof(Color4), hipMemcpyHostToDevice);

    Color4 *pixels_d;
    hipMalloc(&pixels_d, total_pixel*sizeof(Color4));
    hipMemcpy(pixels_d, pixels, total_pixel*sizeof(Color4), hipMemcpyHostToDevice);

    /*int *pre_label_d;
    hipMalloc(&pre_label_d, total_pixel*sizeof(int));
    hipMemcpy(pre_label_d, pre_label, total_pixel*sizeof(int), hipMemcpyHostToDevice);*/

    int *label_d;
    hipMalloc(&label_d, total_pixel*sizeof(int));
    hipMemcpy(label_d, label, total_pixel*sizeof(int), hipMemcpyHostToDevice);

    int *migration_count_d;
    hipMalloc(&migration_count_d, sizeof(int));

    dim3 threadsPerBlock(256);
    dim3 numBlocks((total_pixel + threadsPerBlock.x - 1) / threadsPerBlock.x);
    classifyPointsKernel<<<numBlocks, threadsPerBlock>>>(centroid_d, label_d, pixels_d, migration_count_d, cluster_count, total_pixel);
    hipDeviceSynchronize();

    hipMemcpy(label, label_d, total_pixel*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(migration_count, migration_count_d, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(centroid_d);
    hipFree(pixels_d);
    hipFree(label_d);
    //hipFree(pre_label_d);
    hipFree(migration_count_d);
}

__global__ void updateCentroidKernel(Color4_SUM* labelSum, int* labelCount, Color4* pixels, int* label, int total_pixel, int cluster_count)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize shared memory for label count and sum
    extern __shared__ Color4_SUM sharedLabelSum[];
    //extern __shared__ int sharedLabelCount[];

    if (tid == 0){
        for (int i = 0; i < cluster_count; i++)
        {
            labelSum[i].r = 0;
            labelSum[i].g = 0;
            labelSum[i].b = 0;
            labelSum[i].a = 0;

            labelCount[i] = 0;
        }
    }

    if (threadIdx.x == 0){
        for (int i = 0; i < cluster_count; i++)
        {
            sharedLabelSum[i].r = 0;
            sharedLabelSum[i].g = 0;
            sharedLabelSum[i].b = 0;
            sharedLabelSum[i].a = 0;

            //sharedLabelCount[i] = 0;
        }

    }

    __syncthreads();

    // Update shared memory with partial sums and counts
    if (tid < total_pixel)
    {
        int currentLabel = label[tid];
        //atomicAdd(&sharedLabelCount[currentLabel], 1);
        //printf("BlockIdx: %d, sharedLabelCount[0]: %d, sharedLabelSum[0].r: %d\n", blockIdx.x, sharedLabelCount[0], sharedLabelSum[0].r);
        atomicAdd(reinterpret_cast<unsigned int*>(&sharedLabelSum[currentLabel].r), static_cast<unsigned int>(pixels[tid].r));
        atomicAdd(reinterpret_cast<unsigned int*>(&sharedLabelSum[currentLabel].g), static_cast<unsigned int>(pixels[tid].g));
        atomicAdd(reinterpret_cast<unsigned int*>(&sharedLabelSum[currentLabel].b), static_cast<unsigned int>(pixels[tid].b));
        atomicAdd(reinterpret_cast<unsigned int*>(&sharedLabelSum[currentLabel].a), 1);
    }

    __syncthreads();

    // Update global memory with shared memory values
    if (threadIdx.x == 0)
    {   
        for(int i = 0; i < cluster_count; i++)
        {
            //atomicAdd(&labelCount[i], sharedLabelCount[i]);
            //printf("slc: %d ", sharedLabelCount[i]);
            atomicAdd(reinterpret_cast<unsigned int*>(&labelSum[i].r), static_cast<unsigned int>(sharedLabelSum[i].r));
            atomicAdd(reinterpret_cast<unsigned int*>(&labelSum[i].g), static_cast<unsigned int>(sharedLabelSum[i].g));
            atomicAdd(reinterpret_cast<unsigned int*>(&labelSum[i].b), static_cast<unsigned int>(sharedLabelSum[i].b));
            atomicAdd(reinterpret_cast<unsigned int*>(&labelSum[i].a), static_cast<unsigned int>(sharedLabelSum[i].a));
        }
    }
}

void host_update_centroid(Color4 *centroid, int *label, Color4 *pixels, int cluster_count, int total_pixel)
{
    int *label_d;
    hipMalloc(&label_d, total_pixel*sizeof(int));
    hipMemcpy(label_d, label, total_pixel*sizeof(int), hipMemcpyHostToDevice);

    Color4 *pixels_d;
    hipMalloc(&pixels_d, total_pixel*sizeof(Color4));
    hipMemcpy(pixels_d, pixels, total_pixel*sizeof(Color4), hipMemcpyHostToDevice);

    Color4_SUM *label_sum = (Color4_SUM *)malloc(cluster_count * sizeof(Color4_SUM));
    Color4_SUM *label_sum_d;
    hipMalloc(&label_sum_d, cluster_count*sizeof(Color4_SUM));

    int *label_count = (int *)malloc(cluster_count * sizeof(int));
    int *label_count_d;
    hipMalloc(&label_count_d, cluster_count*sizeof(int));

    // Calculate each number of each label
    dim3 threadsPerBlock(256);
    dim3 numBlocks((total_pixel + threadsPerBlock.x - 1) / threadsPerBlock.x);
    int sharedMemorySize = sizeof(Color4_SUM) * cluster_count + sizeof(int) * + cluster_count;
    updateCentroidKernel<<<numBlocks, threadsPerBlock, sharedMemorySize>>>(label_sum_d, label_count_d, pixels_d, label_d, total_pixel, cluster_count);
    hipDeviceSynchronize();
    
    hipMemcpy(label_sum, label_sum_d, cluster_count*sizeof(Color4_SUM), hipMemcpyDeviceToHost);
    //hipMemcpy(label_count, label_count_d, cluster_count*sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0;  i < cluster_count; i++)
        label_count[i] = label_sum[i].a;

    //printf("%d\n", total_pixel);
    for (int i = 0; i < cluster_count; i++)
    { 
        printf("%d ", label_count[i]);
        printf("%ld ", label_sum[i].r);
        printf("%ld ", label_sum[i].g);
        printf("%ld ", label_sum[i].b);
        printf("%ld \n", label_sum[i].a);
    }
    printf("\n");

    for (int i = 0; i < cluster_count; i++)
    {
        // printf("%ld",label_sum[i].r/label_count[i]);
        if (label_sum[i].r != 0 && label_count[i] != 0)
            centroid[i].r = label_sum[i].r / label_count[i];
        if (label_sum[i].g != 0 && label_count[i] != 0)
            centroid[i].g = label_sum[i].g / label_count[i];
        if (label_sum[i].b != 0 && label_count[i] != 0)
            centroid[i].b = label_sum[i].b / label_count[i];
        if (label_sum[i].a != 0 && label_count[i] != 0)
            centroid[i].a = 0;
    }
    for(int i = 0; i < cluster_count; i++){
        printf("%d ", centroid[i].r);
        printf("%d ",centroid[i].g);
        printf("%d ",centroid[i].b);
        printf("\n");
    }

    free(label_sum);
    free(label_count);
    hipFree(label_sum_d);
    hipFree(label_count_d);
    hipFree(label_d);
    hipFree(pixels_d);
}